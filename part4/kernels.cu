#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>


__global__ void gpu_HeatReduce (float *h, float *g, int N) {
    // float diff, sum=0.0;
    int j = threadIdx.x + blockDim.x * blockIdx.x + 1;
    int i = threadIdx.y + blockDim.y * blockIdx.y + 1;
    if (i < N - 1 && j < N - 1) {
        g[i*N + j]= 0.25 * ( h[ i*N + (j-1) ]+  // left
                             h[ i*N     + (j+1) ]+  // right
                             h[ (i-1)*N + j     ]+  // top
                             h[ (i+1)*N + j     ]); // bottom
        // diff = g[i*N + j] - h[i*N + j];
        // sum += diff * diff;
    }
}

__global__ void gpu_Heat (float *h, float *g, int N) {
	int j = threadIdx.x + blockDim.x * blockIdx.x + 1;
	int i = threadIdx.y + blockDim.y * blockIdx.y + 1;
	if (i < N - 1 && j < N - 1) {
		g[i*N + j]= 0.25 * ( h[ i*N + (j-1) ]+  // left
				     h[ i*N     + (j+1) ]+  // right
				     h[ (i-1)*N + j     ]+  // top
				     h[ (i+1)*N + j     ]); // bottom
	}
}

__global__ void gpu_Diff(float *h, float *g, int N) {
	float diff;
	int j = threadIdx.x + blockDim.x * blockIdx.x + 1;
	int i = threadIdx.y + blockDim.y * blockIdx.y + 1;
	if (i < N - 1 && j < N - 1) {
		diff = g[i*N + j] - h[i*N + j];
		h[i*N + j] = diff * diff; 
	}
}

__global__ void gpu_Reduce(float *g, int N, int scale) {
  __shared__ float sdata[256];
	int j = (threadIdx.x + blockDim.x * blockIdx.x) * scale + 1;
	int i = (threadIdx.y + blockDim.y * blockIdx.y) * scale + 1;
	int sdata_size = blockDim.x*blockDim.y;
	int sdata_index = threadIdx.x + threadIdx.y * blockDim.x;
    sdata[sdata_index] = i < N && j < N ? g[i * N + j]: 0.0;
    __syncthreads();
    // do reduction in shared mem
    for (int s=1; s < sdata_size; s *=2) {
        int index = 2 * s * sdata_index;
        if (index < sdata_size) {
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (sdata_index == 0) g[i * N + j] = sdata[0];
}

__global__ void gpu_Reduce_Atomic(float *g, int N) {
	__shared__ float sdata[256];
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[threadIdx.x] = i < N ? g[i]: 0.0;

	__syncthreads();
	// do reduction in shared mem
	for (int s=1; s < blockDim.x; s *=2)
	{
		int index = 2 * s * threadIdx.x;;

		if (index < blockDim.x)
		{
			sdata[index] += sdata[index + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (threadIdx.x == 0)
		atomicAdd(g, sdata[0]);
}
